#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "common.h"
#include <hip/hip_runtime.h>

__global__ void det_calc_onDevice(double *matrix,double * results, int n);
void det_calc_onHost(int id, double *matrix, int n);

int main(int argc, char **argv) {

    printf("%s Starting...\n", argv[0]);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    char *files[5];
    int input = 0;
    int num_files = 0;
    double *h_matrix;
    double *d_matrix;
    double *h_results;
    double *d_results;

    while (input != -1) {
        input = getopt(argc, argv, "t:i:");
        if (input == 'i')
            files[num_files++] = optarg;
    }

    for (int i=0; i<num_files; i++){
        FILE * f;
        f = fopen(files[i], "rb");

        int a;
        int n_matrices;
        int order;
        a=fread(&n_matrices, 4, 1, f);
        a=fread(&order, 4, 1, f);

        //read and store on host each matrix
        h_matrix = (double*) malloc(n_matrices*order*order*sizeof(double));
        a=fread(h_matrix, 8, n_matrices*order*order, f);

        // allocate space for Host results
        h_results = (double *)malloc(sizeof(double) * n_matrices * order);

        input = a;
        
        for (int i=0; i<n_matrices;i++){
                h_results[i] = 1;
        }

        printf("\n\nProcessing file: %s\n", argv[num_files + 1]);

        // prepare launching grid
        dim3 grid, block;
        grid.x = n_matrices;
        grid.y = 1;
        grid.z = 1;
        block.x = order;
        block.y = 1;
        block.z = 1;


        //malloc device memory
        CHECK(hipMalloc(&d_results, 8*n_matrices*order));
        CHECK(hipMalloc(&d_matrix, n_matrices*order*order*8));

        // transfer data from host to device
        CHECK(hipMemcpy(d_matrix, h_matrix, n_matrices*order*order*8, hipMemcpyHostToDevice));
        CHECK(hipMemcpy(d_results, h_results, n_matrices*order*8, hipMemcpyHostToDevice));

        double iStart = seconds();
        // det_calc_onHost(n_matrices, h_matrix, order);
        // double iElapsHost = seconds() - iStart;

        // free matrices on Host
        free(h_matrix);

        iStart = seconds();
        det_calc_onDevice<<< grid, block >>>(d_matrix, d_results, order);
        CHECK(hipDeviceSynchronize());
        double iElaps = seconds() - iStart;

        // free matrices on Device
        hipFree(d_matrix);

        // copy kernel result back to host side
        hipMemcpy(h_results, d_results, sizeof(double) * n_matrices, hipMemcpyDeviceToHost);

        // free device global memory
        hipFree(d_results);

        printf("\n\n\n\n");

        for (int j = 0; j < n_matrices; j++) {
            printf("Processing matrix %d\n", j + 1);
            printf("The determinant is %.3e\n", h_results[j]);
        }
        // printf("Time elapsed on det_calc_onHost >>> %f\n", iElapsHost);
        printf("Time elapsed on det_calc_onDevice >>> %f\n", iElaps);
        
        fclose (f);
    }

    return 0;


}


__global__ void det_calc_onDevice(double *matrix,double* results, int n)
{

    int m_idx = blockIdx.x+gridDim.x*blockIdx.y+gridDim.x*gridDim.y*blockIdx.z;

    int c_idx = threadIdx.x+blockDim.x*threadIdx.y+blockDim.x*blockDim.y*threadIdx.z;

    int mat = m_idx*n*n;
    int col = mat + c_idx;

    for(int i=0; i<n; i++){

            if(c_idx < i) continue;

            if(threadIdx.x == i){
                    results[m_idx] *= matrix[mat+i + i*n];
                    continue;
            }
            for(int j = i+1; j< n; j++)
                        matrix[col + j*n] -= matrix[mat+i + j*n] * matrix[col + i*n] / matrix[mat+i + i*n];

        __syncthreads();
    }

}


void det_calc_onHost(int n_mat, double *matrix, int n)
{

    for(int m_idx=0; m_idx < n_mat; m_idx++){

	int mat = m_idx * n * n;

        double det = 1;


        for (int c_idx = 0; c_idx < n; c_idx++){

            int col = mat + c_idx;

            for(int i=0; i<n; i++){

                 if(c_idx < i) continue;

                 if(c_idx == i){
                    det *= matrix[mat+i + i*n];
                    continue;
                }

                 for(int j = i+1; j< n; j++)
                        matrix[col + j*n] -= matrix[mat+i + j*n] * matrix[col + i*n] / matrix[mat+i + i*n];

            }

        }

        //printf("Processing matrix %d\n", m_idx+1);
        //printf("The determinant is %.3e\n", det);


    }
}